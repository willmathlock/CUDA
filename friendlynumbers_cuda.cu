#include "hip/hip_runtime.h"
/*
152096 - William Matheus
Friendly Numbers
Programacao Paralela e Distribuida
CUDA - 2019/2 - UPF
*/


#include <stdio.h>
#include <hip/hip_runtime.h>


#define THREADSPERBLOCK 1024
#define NUMBEROFBLOCK 	3

__device__ void gcd ( int a, int b, int *result){
	int c;	
	while ( a != 0 ) {
     	c = a; 
     	a = b % a;
		b = c;
	}
    *result = b;
}

__global__ void FuncNumDem(long int* device_num, long int* device_den, long int start, long int end, int size, int inc)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x + inc;
	int result;
	long int factor, ii, sum, done, n;

	if (i < size) {
		ii = i - start;
		sum = 1 + i;
		done = i;
		factor = 2;
		while (factor < done) {
			if ((i % factor) == 0) {
				sum += (factor + (i / factor));
				if ((done = i / factor) == factor)
					sum -= factor;
			}
			factor++;
		}
		device_num[ii] = sum;
		device_den[ii] = i;
	 	gcd(device_num[ii], device_den[ii], &result);
		n = result;
		device_num[ii] /= n;
		device_den[ii] /= n;
	}
}

__global__ void FuncSum(long int* device_num, long int* device_den, long int* device_vet, int size, int incremento)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x + incremento;
	int j;

	if (i < size) {
		for (j = i + 1; j < size; j++) {
			if ((device_num[i] == device_num[j]) && (device_den[i] == device_den[j]))
				device_vet[i]++;
		}
	}
}

void friendly_numbers(long int start, long int end) {
	hipSetDevice(0);

	int number_grid, c=0, i;
	long int *device_num, *device_den, *device_vet;

	long int last = end - start + 1;
	size_t size = last * sizeof(long int);
	number_grid = last / (NUMBEROFBLOCK * THREADSPERBLOCK)+1;

	long int *num;
	long int *den;
	long int *vet;

	num = (long int*) malloc(size);
	den = (long int*) malloc(size);
	vet = (long int*) malloc(size);

	hipMalloc((void**)&device_num, size);
	hipMalloc((void**)&device_den, size);
	hipMalloc((void**)&device_vet, size);
	
	for (i = 0; i < last; i++) {
		vet[i] = 0;
	}

	hipMemcpy(cuda_num, num, size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_den, den, size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_vet, vet, size, hipMemcpyHostToDevice);

	int incremento = 0;

	for (i = 0; i < number_grid; i++) {
		FuncNumDem<<<NUMBEROFBLOCK, THREADSPERBLOCK>>>(cuda_num, cuda_den, start, end, last, incremento);
		incremento += NUMBEROFBLOCK * THREADSPERBLOCK;
	}
	
	incremento = 0;

	for (i = 0; i < number_grid; i++) {
		FuncSum<<<NUMBEROFBLOCK, THREADSPERBLOCK>>>(cuda_num, cuda_den, cuda_vet, last, incremento);
		incremento += NUMBEROFBLOCK * THREADSPERBLOCK;
	}

	hipMemcpy(vet, cuda_vet, size, hipMemcpyDeviceToHost);

	for (i = 0; i < last; i++) {
		c += vet[i];
	}

	printf("Found %d pairs of mutually friendly numbers\n", c);

	free(num);
	free(den);
	free(vet);

	hipFree(cuda_num);
	hipFree(cuda_den);
	hipFree(cuda_vet);
}

int main(int argc, char **argv) {
	long int start;
	long int end;

	if (argc != 3){
		printf("Wrong number of arguments\n");
		return EXIT_FAILURE;
	}

	start = atoi(argv[1]);
	end = atoi(argv[2]);

	printf("Number %ld to %ld\n", start, end);
	friendly_numbers(start, end);

	return EXIT_SUCCESS;
}
