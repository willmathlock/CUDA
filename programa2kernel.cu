/*
152096 - William Matheus
Friendly Numbers
Programacao Paralela e Distribuida
CUDA - 2019/2 - UPF
Programa 2 - Kernel
*/


#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ void divisionRemainder(int num, int den, int *divisionRemainder){
	*divisionRemainder = num / den;
	*divisionRemainder = num - (den * *divisionRemainder);

}

__device__ void gcd ( int a, int b, int *result){
	int c, resto;	
	while ( a != 0 ) {
     	c = a; 
     	divisionRemainder(b, a, &resto);
     	a = resto;
		b = c;
	}
    *result = b;
}


__global__ void numDem(long int *device_num, long int *device_den, long int start, long int end, int size, int inc)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x + inc;
	int result, resto;
	long int factor, ii, sum, done, n;

	if (i < size) {
		ii = i - start;
		sum = 1 + i;
		done = i;
		factor = 2;
		while (factor < done) {
			divisionRemainder(i, factor, &resto);
			if (resto== 0) {
				sum += (factor + (i / factor));
				//doneFactor(i, factor, &done);
				if ((done = i / factor) == factor)
					sum -= factor;
			}
			factor++;
		}
		device_num[ii] = sum;
		device_den[ii] = i;
	 	gcd(device_num[ii], device_den[ii], &result);
		n = result;
		device_num[ii] /= n;
		device_den[ii] /= n;
	}
}

__global__ void sum(long int* device_num, long int* device_den, long int* device_vet, int size, int x)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x + x;
	int j;

	if (i < size) {
		for (j = i + 1; j < size; j++) {
			if ((device_num[i] == device_num[j]) && (device_den[i] == device_den[j]))
				device_vet[i]++;
		}
	}
}

void friendly_numbers(long int start, long int end) {
	hipSetDevice(0);


	int deviceId;
	int number_grid, c=0, i;
	long int *device_num, *device_den, *device_vet;

	struct hipDeviceProp_t prop;

	hipGetDevice(&deviceId);
	hipGetDeviceProperties(&prop, deviceId);


	long int last = end - start + 1;
	size_t size = last * sizeof(long int);

	int nBlocks = 4*((prop.maxThreadsPerMultiProcessor * prop.multiProcessorCount)/prop.maxThreadsPerBlock);	
	int threads = prop.maxThreadsPerBlock;

	number_grid = last / (nBlocks * threads)+1;

	long int *num;
	long int *den;
	long int *vet;

	num = (long int*) malloc(size);
	den = (long int*) malloc(size);
	vet = (long int*) malloc(size);

	hipMalloc((void**)&device_num, size);
	hipMalloc((void**)&device_den, size);
	hipMalloc((void**)&device_vet, size);
	
	for (i = 0; i < last; i++) {
		vet[i] = 0;
	}

	hipMemcpy(device_num, num, size, hipMemcpyHostToDevice);
	hipMemcpy(device_den, den, size, hipMemcpyHostToDevice);
	hipMemcpy(device_vet, vet, size, hipMemcpyHostToDevice);

	int x = 0;

	for (i = 0; i < number_grid; i++) {
		numDem<<<nBlocks, threads>>>(device_num, device_den, start, end, last, x);
		x += nBlocks * threads;
	}
	
	x = 0;

	for (i = 0; i < number_grid; i++) {
		sum<<<nBlocks, threads>>>(device_num, device_den, device_vet, last, x);
		x += nBlocks * threads;;
	}

	hipMemcpy(vet, device_vet, size, hipMemcpyDeviceToHost);

	for (i = 0; i < last; i++) {
		c += vet[i];
	}

	printf("Found %d pairs of mutually friendly numbers\n", c);

	free(num);
	free(den);
	free(vet);

	hipFree(device_num);
	hipFree(device_den);
	hipFree(device_vet);
}

int main(int argc, char **argv) {
	long int start;
	long int end;

	if (argc != 3){
		printf("Wrong number of arguments\n");
		return EXIT_FAILURE;
	}

	start = atoi(argv[1]);
	end = atoi(argv[2]);

	printf("Number %ld to %ld\n", start, end);
	friendly_numbers(start, end);

	return EXIT_SUCCESS;
}
